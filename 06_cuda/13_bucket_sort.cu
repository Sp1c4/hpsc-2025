
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <vector>

__global__ void countKeys(int *key, int n, int range, int *bucket) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
      atomicAdd(&bucket[key[i]], 1);
  }
}

__global__ void reconstructArray(int *key, int n, int range, int *bucket) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < range) {
      int start_index = 0;
      for (int j = 0; j < i; ++j) {
          start_index += bucket[j];
      }
      int count = bucket[i];
      for (int k = 0; k < count; ++k)
      {
           int index = start_index + k;
           if(index < n){
              key[index] = i;
           }
      }
  }
}

int main() {
  int n = 50;
  int range = 5;
  std::vector<int> key(n);
  int *bucket_device;
  int *key_device;
  hipMallocManaged(&key_device, n * sizeof(int));
  hipMallocManaged(&bucket_device, range * sizeof(int));
  for (int i=0; i<n; i++) {
    key[i] = rand() % range;
    printf("%d ",key[i]);
  }
  for (int j=0; j<range; j++) {
    bucket_device[j] = 0;
  }
  printf("\n");

  hipMemcpy(key_device, key.data(), n * sizeof(int), hipMemcpyHostToDevice);
  countKeys<<<1, 64>>>(key_device, n, range, bucket_device);
  hipDeviceSynchronize();

  reconstructArray<<<1, 64>>>(key_device, n, range, bucket_device);
  hipDeviceSynchronize();

  hipMemcpy(key.data(), key_device, n * sizeof(int), hipMemcpyDeviceToHost);
  for (int i=0; i<n; i++) {
    printf("%d ",key[i]);
  }
  printf("\n");
  hipFree(key_device);
  hipFree(bucket_device);
}
